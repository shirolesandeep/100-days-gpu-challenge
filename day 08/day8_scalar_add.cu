#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void scalarAdd(float *x, float *y, float a, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) y[idx] = x[idx] + a;
}

int main() {
    int n = 1024;
    size_t size = n * sizeof(float);
    float a = 3.0f;

    float *h_x = (float*)malloc(size);
    float *h_y = (float*)malloc(size);

    for (int i = 0; i < n; i++) {
        h_x[i] = 2.0f;
    }

    float *d_x, *d_y;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    scalarAdd<<<blocks, threads>>>(d_x, d_y, a, n);
    hipDeviceSynchronize();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    printf("Day 8 CUDA C: y[0] = %.1f (expected 5.0)\n", h_y[0]);

    hipFree(d_x); hipFree(d_y);
    free(h_x); free(h_y);
    return 0;
}
