#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// Error checking macro
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in %s:%d: %s\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while(0)

// Matrix multiplication kernel
__global__ void matrixMulKernel(float *A, float *B, float *C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N && col < N) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

int main() {
    const int N = 1024; // Matrix size (N x N)
    const int blockSize = 16; // Thread block size
    int numGPUs;
    
    // Get number of available GPUs
    CUDA_CHECK(hipGetDeviceCount(&numGPUs));
    printf("Number of GPUs: %d\n", numGPUs);
    
    // Matrix size per GPU
    int chunkSize = N / numGPUs;
    
    // Host matrices
    float *h_A, *h_B, *h_C;
    size_t size = N * N * sizeof(float);
    
    h_A = (float*)malloc(size);
    h_B = (float*)malloc(size);
    h_C = (float*)malloc(size);
    
    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        h_A[i] = rand() / (float)RAND_MAX;
        h_B[i] = rand() / (float)RAND_MAX;
        h_C[i] = 0.0f;
    }
    
    // Device arrays and streams
    float **d_A = (float**)malloc(numGPUs * sizeof(float*));
    float **d_B = (float**)malloc(numGPUs * sizeof(float*));
    float **d_C = (float**)malloc(numGPUs * sizeof(float*));
    hipStream_t *streams = (hipStream_t*)malloc(numGPUs * sizeof(hipStream_t));
    
    // Timing
    hipEvent_t start, stop;
    CUDA crocodiles_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    // Initialize devices and streams
    for (int i = 0; i < numGPUs; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipMalloc(&d_A[i], chunkSize * N * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_B[i], N * N * sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_C[i], chunkSize * N * sizeof(float)));
        CUDA_CHECK(hipStreamCreate(&streams[i]));
    }
    
    // Start timing
    CUDA_CHECK(hipEventRecord(start));
    
    // Launch kernels on each GPU
    dim3 threadsPerBlock(blockSize, blockSize);
    dim3 blocksPerGrid((N + blockSize - 1) / blockSize, 
                      (chunkSize + blockSize - 1) / blockSize);
    
    for (int i = 0; i < numGPUs; i++) {
        CUDA_CHECK(hipSetDevice(i));
        
        // Copy input data to device
        CUDA_CHECK(hipMemcpyAsync(d_A[i], h_A + i * chunkSize * N,
                                 chunkSize * N * sizeof(float),
                                 hipMemcpyHostToDevice, streams[i]));
        CUDA_CHECK(hipMemcpyAsync(d_B[i], h_B,
                                 N * N * sizeof(float),
                                 hipMemcpyHostToDevice, streams[i]));
        
        // Launch kernel
        matrixMulKernel<<<blocksPerGrid, threadsPerBlock, 0, streams[i]>>>
                       (d_A[i], d_B[i], d_C[i], N);
        
        // Copy results back
        CUDA_CHECK(hipMemcpyAsync(h_C + i * chunkSize * N, d_C[i],
                                 chunkSize * N * sizeof(float),
                                 hipMemcpyDeviceToHost, streams[i]));
    }
    
    // Synchronize all streams
    for (int i = 0; i < numGPUs; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipStreamSynchronize(streams[i]));
    }
    
    // Stop timing
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    printf("Execution time: %f ms\n", milliseconds);
    
    // Verify results
    float maxError = 0.0f;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            float ref = 0.0f;
            for (int k = 0; k < N; k++) {
                ref += h_A[i * N + k] * h_B[k * N + j];
            }
            float diff = fabs(h_C[i * N + j] - ref);
            maxError = fmax(maxError, diff);
        }
    }
    printf("Maximum error: %f\n", maxError);
    
    // Cleanup
    for (int i = 0; i < numGPUs; i++) {
        CUDA_CHECK(hipSetDevice(i));
        CUDA_CHECK(hipFree(d_A[i]));
        CUDA_CHECK(hipFree(d_B[i]));
        CUDA_CHECK(hipFree(d_C[i]));
        CUDA_CHECK(hipStreamDestroy(streams[i]));
    }
    
    free(d_A);
    free(d_B);
    free(d_C);
    free(streams);
    free(h_A);
    free(h_B);
    free(h_C);
    
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    
    return 0;
}