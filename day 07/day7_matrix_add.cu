#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixAdd(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idx = row * n + col;
    if (row < n && col < n) C[idx] = A[idx] + B[idx];
}

int main() {
    int n = 32;
    size_t size = n * n * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < n * n; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
    matrixAdd<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Day 7 CUDA C: C[0] = %.1f (expected 3.0)\n", h_C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
