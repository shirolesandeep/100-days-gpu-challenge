#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void matrixMult(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;
    if (row < n && col < n) {
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int n = 32;  // Small size for simplicity
    size_t size = n * n * sizeof(float);

    float *h_A = (float*)malloc(size);
    float *h_B = (float*)malloc(size);
    float *h_C = (float*)malloc(size);

    for (int i = 0; i < n * n; i++) {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((n + threads.x - 1) / threads.x, (n + threads.y - 1) / threads.y);
    matrixMult<<<blocks, threads>>>(d_A, d_B, d_C, n);
    hipDeviceSynchronize();

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    printf("Day 2 CUDA C: C[0] = %.1f (expected 64.0)\n", h_C[0]);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    free(h_A); free(h_B); free(h_C);
    return 0;
}
