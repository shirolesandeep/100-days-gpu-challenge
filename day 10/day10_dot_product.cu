#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dotProduct(float *x, float *y, float *out, int n) {
    extern __shared__ float sdata[];
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (idx < n) ? x[idx] * y[idx] : 0.0f;
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) sdata[tid] += sdata[tid + s];
        __syncthreads();
    }

    if (tid == 0) out[blockIdx.x] = sdata[0];
}

int main() {
    int n = 1024;
    size_t size = n * sizeof(float);

    float *h_x = (float*)malloc(size);
    float *h_y = (float*)malloc(size);
    float *h_out = (float*)malloc(sizeof(float));

    for (int i = 0; i < n; i++) {
        h_x[i] = 2.0f;
        h_y[i] = 3.0f;
    }

    float *d_x, *d_y, *d_out;
    hipMalloc(&d_x, size);
    hipMalloc(&d_y, size);
    hipMalloc(&d_out, sizeof(float));

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, h_y, size, hipMemcpyHostToDevice);

    int threads = 256;
    int blocks = (n + threads - 1) / threads;
    dotProduct<<<blocks, threads, threads * sizeof(float)>>>(d_x, d_y, d_out, n);
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, sizeof(float), hipMemcpyDeviceToHost);

    printf("Day 10 CUDA C: Dot product = %.1f (expected 6144.0)\n", *h_out);

    hipFree(d_x); hipFree(d_y); hipFree(d_out);
    free(h_x); free(h_y); free(h_out);
    return 0;
}
