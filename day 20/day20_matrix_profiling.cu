#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <stdio.h>

#define CHECK_CUDA(call) { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        exit(1); \
    } \
}

#define CHECK_CUBLAS(call) { \
    hipblasStatus_t stat = call; \
    if (stat != HIPBLAS_STATUS_SUCCESS) { \
        printf("cuBLAS error at %s:%d\n", __FILE__, __LINE__); \
        exit(1); \
    } \
}

void profileMatrixMul(int m, int n, int k, float* A, float* B, float* C) {
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));

    float *d_A, *d_B, *d_C;
    CHECK_CUDA(hipMalloc(&d_A, m * k * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_B, k * n * sizeof(float)));
    CHECK_CUDA(hipMalloc(&d_C, m * n * sizeof(float)));

    CHECK_CUDA(hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));

    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Warm-up run
    CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m));

    // Profile
    CHECK_CUDA(hipEventRecord(start));
    for (int i = 0; i < 10; i++) {  // Average over 10 runs
        CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, d_A, m, d_B, k, &beta, d_C, m));
    }
    CHECK_CUDA(hipEventRecord(stop));
    CHECK_CUDA(hipEventSynchronize(stop));

    float milliseconds = 0;
    CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
    printf("cuBLAS Matrix Mul (%dx%d x %dx%d): %f ms (avg over 10 runs)\n", m, k, k, n, milliseconds / 10);

    CHECK_CUDA(hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost));

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
}

int main() {
    int m = 1024, n = 1024, k = 1024;
    float* A = (float*)malloc(m * k * sizeof(float));
    float* B = (float*)malloc(k * n * sizeof(float));
    float* C = (float*)malloc(m * n * sizeof(float));

    // Initialize matrices (simple example)
    for (int i = 0; i < m * k; i++) A[i] = (float)(i % 10);
    for (int i = 0; i < k * n; i++) B[i] = (float)(i % 5);

    profileMatrixMul(m, n, k, A, B, C);

    free(A); free(B); free(C);
    return 0;
}
