#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512  // Matrix size (N x N)
#define TILE_SIZE 32  // Tile size for shared memory

// CUDA kernel for tiled matrix multiplication using shared memory
__global__ void matrixMulTiled(float *A, float *B, float *C, int n) {
    // Shared memory for tiles of A and B
    __shared__ float tileA[TILE_SIZE][TILE_SIZE];
    __shared__ float tileB[TILE_SIZE][TILE_SIZE];
    
    int row = blockIdx.y * TILE_SIZE + threadIdx.y;
    int col = blockIdx.x * TILE_SIZE + threadIdx.x;
    float sum = 0.0f;
    
    // Loop over tiles
    for (int t = 0; t < (n + TILE_SIZE - 1) / TILE_SIZE; t++) {
        // Load tiles into shared memory
        int a_col = t * TILE_SIZE + threadIdx.x;
        int b_row = t * TILE_SIZE + threadIdx.y;
        
        tileA[threadIdx.y][threadIdx.x] = (row < n && a_col < n) ? A[row * n + a_col] : 0.0f;
        tileB[threadIdx.y][threadIdx.x] = (b_row < n && col < n) ? B[b_row * n + col] : 0.0f;
        __syncthreads();
        
        // Compute partial sum for this tile
        for (int k = 0; k < TILE_SIZE; k++) {
            sum += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
        }
        __syncthreads();
    }
    
    if (row < n && col < n) {
        C[row * n + col] = sum;
    }
}

// Helper function to check CUDA errors
void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        printf("%s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    int n = N;
    size_t bytes = n * n * sizeof(float);
    
    // Host matrices
    float *h_A, *h_B, *h_C;
    h_A = (float*)malloc(bytes);
    h_B = (float*)malloc(bytes);
    h_C = (float*)malloc(bytes);
    
    // Initialize matrices
    for (int i = 0; i < n * n; i++) {
        h_A[i] = (float)(i % 10);  // Simple pattern
        h_B[i] = (float)((i + 1) % 10);
    }
    
    // Device matrices
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc(&d_A, bytes), "CUDA malloc d_A failed");
    checkCudaError(hipMalloc(&d_B, bytes), "CUDA malloc d_B failed");
    checkCudaError(hipMalloc(&d_C, bytes), "CUDA malloc d_C failed");
    
    // Copy data to device
    checkCudaError(hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice), "CUDA memcpy to d_A failed");
    checkCudaError(hipMemcpy(d_B, h_B, bytes, hipMemcpyHostToDevice), "CUDA memcpy to d_B failed");
    
    // Configure kernel launch
    dim3 threadsPerBlock(TILE_SIZE, TILE_SIZE);
    dim3 blocksPerGrid((n + TILE_SIZE - 1) / TILE_SIZE, (n + TILE_SIZE - 1) / TILE_SIZE);
    
    // Launch kernel
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start);
    matrixMulTiled<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);
    hipEventRecord(stop);
    
    checkCudaError(hipGetLastError(), "Kernel launch failed");
    
    // Copy result back to host
    checkCudaError(hipMemcpy(h_C, d_C, bytes, hipMemcpyDeviceToHost), "CUDA memcpy to host failed");
    
    // Print execution time
    float milliseconds = 0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %.4f ms\n", milliseconds);
    
    // Verify results (print a few elements)
    printf("Sample results (top-left 3x3):\n");
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            printf("%.1f ", h_C[i * n + j]);
        }
        printf("\n");
    }
    
    // Cleanup
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);
    
    return 0;
}
